#include <stdio.h>
#include <hip/hip_runtime.h>

bool init_cuda()
{
    int count;

    hipGetDeviceCount(&count);
    if(0 == count){
        fprintf(stderr,"There is no device\n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++){
        hipDeviceProp_t prop;
        if(hipSuccess == hipGetDeviceProperties(&prop,i)){
            if(prop.major >= 1){
                break;
            }
        }
    }

    if(i == count){
        fprintf(stderr,"There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

int main()
{
    if(!init_cuda()){
        return 0;
    }

    printf("CUDA initialize.\n");

    return 0;
}
